#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////////////////
// 
// File: CUDASilhouetteFinding.cu
// 
// Author: Ren Yifei, yfren@cs.hku.hk
//
// Desc: CUDA kernel code for GP-GPU processing in parallel
//
//////////////////////////////////////////////////////////////////////////////////////////////////

#include "CUDASilhouetteFinding.h"
#include "CUDADataStructure.h"

int	h_curMaxIndiceNum = 0;
int h_curMaxVertexNum = 0;
int h_curMaxSilNum = 0;

__device__ MeshVertex* 	d_meshVertex = NULL;
__device__ WORD*		d_indices = NULL;
__device__ DWORD*		d_adjBuffer = NULL;
__device__ int*			d_maxIndiceNum = NULL;
__device__ int*			d_silNum = NULL;

__device__ D3DXMATRIX*		d_matrixWorldView  = NULL;
__device__ D3DXMATRIX*		d_matrixProj = NULL;

__device__ D3DXVECTOR3*		d_candidateSilhouetteVertex = NULL;

//�������Σ���һ�α�ʾ�Ƿ�sil,��СindiceNum/2,�ڶ��ξͱ�ʾ�Ƿ�ɼ���sil, ��Сֻ����ǰ���silNum��
__device__ bool*			d_isSilhouette  = NULL; 

//Silhouette detection
__global__ void findSilhouette(MeshVertex* d_meshVertex,
							   WORD* d_indices, 
							   DWORD* d_adjBuffer, 
							   D3DXMATRIX* d_matrixWorldView,
							   D3DXMATRIX* d_matrixProj,
							   bool*	d_isSilhouette,
							   int* d_maxIndiceNum);

//Invisible silhouette culling
__global__ void cullSilouette(MeshVertex* d_meshVertex,
							 WORD* d_indices,
							 int* d_maxIndiceNum,
							 D3DXVECTOR3* d_candidateSilhouetteVertex,
							 int* d_silNum,
							 bool*	d_isSilhouette,
							 D3DXMATRIX* d_matrixWorldView);


//Projection transform from 3D tO 2D viewport
__global__ void projTransform(D3DXVECTOR3*  d_meshVertexProj,
							  int*			d_silNum,
							  D3DXMATRIX*	d_matrixWorldView,
							  D3DXMATRIX*	d_matrixProj);

//Segment / Triangle crossing testing
__device__ bool segmentIntersectTriangle(const D3DXVECTOR3& orig, 
										 const D3DXVECTOR3& des,
										 const D3DXVECTOR3& v0, 
										 const D3DXVECTOR3& v1, 
										 const D3DXVECTOR3& v2);


//Init
bool cudaInitialization(int indiceNum, int vertexNum)
{	
	hipError_t err = hipSuccess;

	if(vertexNum > h_curMaxVertexNum)
	{
		h_curMaxVertexNum = vertexNum;

		if(d_meshVertex)
			hipFree(d_meshVertex);
		
		err = hipMalloc((void**)&d_meshVertex, vertexNum * sizeof(MeshVertex));

		if(err != hipSuccess)
			return false;
	}
	
	if(indiceNum > h_curMaxIndiceNum)
	{
		h_curMaxIndiceNum = indiceNum;

		if(d_indices)
			hipFree(d_indices);
		
		err = hipMalloc((void**)&d_indices, indiceNum * sizeof(WORD));

		if(err != hipSuccess)
			return false;

		if(d_adjBuffer)
			hipFree(d_adjBuffer);

		err = hipMalloc((void**)&d_adjBuffer, indiceNum * sizeof(DWORD));

		if(err != hipSuccess)
			return false;

		if(d_maxIndiceNum)
			hipFree(d_maxIndiceNum);

		err = hipMalloc((void**)&d_maxIndiceNum,		sizeof(int));

		if(err != hipSuccess)
			return false;

		if(d_matrixWorldView)
			hipFree(d_matrixWorldView);

		err = hipMalloc((void**)&d_matrixWorldView,	sizeof(D3DXMATRIX));

		if(err != hipSuccess)
			return false;

		if(d_matrixProj)
			hipFree(d_matrixProj);

		err = hipMalloc((void**)&d_matrixProj,	sizeof(D3DXMATRIX));

		if(err != hipSuccess)
			return false;

		if(d_isSilhouette)
			hipFree(d_isSilhouette);

		err = hipMalloc((void**)&d_isSilhouette, indiceNum * sizeof(bool));

		if(err != hipSuccess)
			return false;
	}

	return true;
}

bool cudaProjInit( int silNum )
{
	hipError_t err = hipSuccess;

	if(silNum > h_curMaxSilNum)
	{
		h_curMaxSilNum = silNum;

		if(d_candidateSilhouetteVertex)
			hipFree(d_candidateSilhouetteVertex);

		err = hipMalloc((void**)&d_candidateSilhouetteVertex, silNum * 2 * sizeof(D3DXVECTOR3));

		if(err != hipSuccess)
			return false;

		if(d_silNum)
			hipFree(d_silNum);

		err = hipMalloc((void**)&d_silNum, sizeof(int));

		if(err != hipSuccess)
			return false;
	}

	return true;
}



bool cudaPassDataToGPU( MeshVertex* _meshVertex, WORD* _indices, DWORD* _adjBuffer, 
						D3DXMATRIX* h_matrixWorldView, D3DXMATRIX* h_matrixProj, 
						int h_indiceNum, int h_vertexNum )
{
	if(!cudaInitialization(h_indiceNum, h_vertexNum))
		return false;
	
	hipMemcpy(d_meshVertex, _meshVertex,		h_vertexNum * sizeof(MeshVertex),		hipMemcpyHostToDevice);
	hipMemcpy(d_indices, _indices,				h_indiceNum * sizeof(WORD),				hipMemcpyHostToDevice);
	hipMemcpy(d_adjBuffer, _adjBuffer,			h_indiceNum * sizeof(DWORD),			hipMemcpyHostToDevice);
	hipMemcpy(d_maxIndiceNum, &h_indiceNum,					 sizeof(int),			hipMemcpyHostToDevice);
	hipMemcpy(d_matrixWorldView, h_matrixWorldView,			 sizeof(D3DXMATRIX),	hipMemcpyHostToDevice);
	hipMemcpy(d_matrixProj, h_matrixProj,						 sizeof(D3DXMATRIX),	hipMemcpyHostToDevice);

	return true;
}

bool cudaPassProjVerticesDataToGPU( D3DXVECTOR3* edgeVertices, int h_silNum )
{
	if(!cudaProjInit(h_silNum))
		return false;

	hipMemcpy(d_silNum, &h_silNum, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_candidateSilhouetteVertex, edgeVertices, h_silNum * 2 * sizeof(D3DXVECTOR3), hipMemcpyHostToDevice);

	return true;
}

bool cudaGetDataFromGPU( bool* h_isSilhouette, int silSize )
{
	hipMemcpy(h_isSilhouette, d_isSilhouette,	 silSize * sizeof(bool), hipMemcpyDeviceToHost);
	
	return true;
}

bool cudaGetProjDataFromGPU( D3DXVECTOR3* h_meshProjVertices, int silSize )
{
	hipMemcpy(h_meshProjVertices, d_candidateSilhouetteVertex, silSize * 2 * sizeof(D3DXVECTOR3), hipMemcpyDeviceToHost);

	return true;
}

bool cudaRunKernel(int indiceNum)
{
	int gridNum = (indiceNum / g_BLOCK_SIZE);
	
	if(indiceNum % g_BLOCK_SIZE != 0)
		++gridNum;


	findSilhouette<<< gridNum, g_BLOCK_SIZE>>> (d_meshVertex, d_indices, d_adjBuffer, 
												d_matrixWorldView, d_matrixProj,
												d_isSilhouette, d_maxIndiceNum);

	hipDeviceSynchronize();

	return true;
}


bool cudaRunProjKernel(int silNum)
{
	int gridNum = (silNum * 2 / g_BLOCK_SIZE);

	if( (silNum * 2) % g_BLOCK_SIZE != 0 )
		++gridNum;

	projTransform<<< gridNum, g_BLOCK_SIZE>>> (d_candidateSilhouetteVertex, d_silNum, d_matrixWorldView, d_matrixProj);

	hipDeviceSynchronize();

	return true;
}

bool cudaRunCullKernel(int silNum, int indiceNum)
{
	int maxTriangleNum = indiceNum / 3;
	
	int gridNum = (silNum * maxTriangleNum / g_BLOCK_SIZE);

	if( (silNum * maxTriangleNum) % g_BLOCK_SIZE != 0 )
		++gridNum;

	hipMemset(d_isSilhouette, 1, sizeof(bool)*silNum);

	cullSilouette<<< gridNum, g_BLOCK_SIZE>>> (d_meshVertex, d_indices, d_maxIndiceNum, 
											  d_candidateSilhouetteVertex, d_silNum, d_isSilhouette, 
											  d_matrixWorldView);
	hipDeviceSynchronize();

	return true;
}


__device__  D3DXVECTOR3 crossProduct(const D3DXVECTOR3& m1, const D3DXVECTOR3& m2)
{
	D3DXVECTOR3 ret;

	ret.x = m1.y * m2.z - m1.z * m2.y;
	ret.y = m1.z * m2.x - m1.x * m2.z;
	ret.z = m1.x * m2.y - m1.y * m2.x;

	return ret;
}

__device__ float dotProduct(const D3DXVECTOR3& m1, const D3DXVECTOR3& m2)
{
	float ret = m1.x * m2.x + m1.y * m2.y + m1.z * m2.z;

	return ret;
}

__device__ float length(const D3DXVECTOR3& vec)
{
	return sqrt(vec.x*vec.x + vec.y*vec.y + vec.z*vec.z);
}

__device__ D3DXVECTOR3 normalize(const D3DXVECTOR3& vec)
{
	D3DXVECTOR3 ret = vec;

	float len = length(ret);

	ret.x /= len;
	ret.y /= len;
	ret.z /= len;

	return ret;
}

__device__ D3DXVECTOR3 matrixPntMul(const D3DXVECTOR3& pnt, const D3DXMATRIX* mat)
{
	
	D3DXVECTOR3 ret;
	
	ret.x = mat->m[0][0] * pnt.x + mat->m[1][0] * pnt.y + mat->m[2][0] * pnt.z + mat->m[3][0];
	ret.y = mat->m[0][1] * pnt.x + mat->m[1][1] * pnt.y + mat->m[2][1] * pnt.z + mat->m[3][1];
	ret.z = mat->m[0][2] * pnt.x + mat->m[1][2] * pnt.y + mat->m[2][2] * pnt.z + mat->m[3][2];
	
	float w = mat->m[0][3] * pnt.x + mat->m[1][3] * pnt.y + mat->m[2][3] * pnt.z + mat->m[3][3];

	ret.x /= w;
	ret.y /= w;
	ret.z /= w;

	return ret;
}

__device__ D3DXVECTOR3 matrixVecMul(const D3DXVECTOR3& vec, const D3DXMATRIX* mat)
{

	D3DXVECTOR3 ret;

	ret.x = mat->m[0][0] * vec.x + mat->m[1][0] * vec.y + mat->m[2][0] * vec.z;
	ret.y = mat->m[0][1] * vec.x + mat->m[1][1] * vec.y + mat->m[2][1] * vec.z;
	ret.z = mat->m[0][2] * vec.x + mat->m[1][2] * vec.y + mat->m[2][2] * vec.z;

	return ret;
}


__global__ void findSilhouette(MeshVertex* d_meshVertex,
							   WORD* d_indices, 
							   DWORD* d_adjBuffer, 
							   D3DXMATRIX* d_matrixWorldView,
							   D3DXMATRIX* d_matrixProj,
							   bool*	d_isSilhouette,
							   int* d_maxIndiceNum)
{
	const int idx = blockIdx.x * g_BLOCK_SIZE + threadIdx.x;

	if(idx >= *d_maxIndiceNum)
		return;
	
	const int idxTriangle	  = idx / 3;
	const int idxTriangleBase = idxTriangle * 3;

	const int idxV0				= d_indices[idxTriangleBase];
	const int idxV1				= d_indices[idxTriangleBase + 1];
	const int idxV2				= d_indices[idxTriangleBase + 2];

	const D3DXVECTOR3& posV0	= d_meshVertex[idxV0].position;
	const D3DXVECTOR3& posV1	= d_meshVertex[idxV1].position;
	const D3DXVECTOR3& posV2	= d_meshVertex[idxV2].position;

	const D3DXVECTOR3 vecV0V1	= posV1 - posV0;
	const D3DXVECTOR3 vecV0V2	= posV2 - posV0;

	D3DXVECTOR3 normal1	= crossProduct(vecV0V1, vecV0V2);

	D3DXVECTOR3 normal2;
	const int idxAdjTriangle = d_adjBuffer[idx];
	
	if(idxAdjTriangle != -1)
	{
		const int idxAdjTriangleBase = idxAdjTriangle * 3;

		const int idxAdjV0			= d_indices[idxAdjTriangleBase];
		const int idxAdjV1			= d_indices[idxAdjTriangleBase + 1];
		const int idxAdjV2			= d_indices[idxAdjTriangleBase + 2];

		const D3DXVECTOR3& posAdjV0	= d_meshVertex[idxAdjV0].position;
		const D3DXVECTOR3& posAdjV1	= d_meshVertex[idxAdjV1].position;
		const D3DXVECTOR3& posAdjV2	= d_meshVertex[idxAdjV2].position;

		const D3DXVECTOR3 vecAdjV0V1	= posAdjV1 - posAdjV0;
		const D3DXVECTOR3 vecAdjV0V2	= posAdjV2 - posAdjV0;

		normal2 = crossProduct(vecAdjV0V1, vecAdjV0V2);
	}
	else
	{
		normal2 = -normal1;
	}

	D3DXVECTOR3 eyeToVertex = matrixPntMul(posV0, d_matrixWorldView);

	normal1 = matrixVecMul(normal1, d_matrixWorldView);
	normal2 = matrixVecMul(normal2, d_matrixWorldView);

	float dot1 = dotProduct(normal1, eyeToVertex);
	float dot2 = dotProduct(normal2, eyeToVertex);
	
	if(dot1 * dot2 < 0.0f)
	{
		//It's a silhouette
		d_isSilhouette[idx] = true;
	}
	else
	{
		d_isSilhouette[idx] = false;
	}
}

__global__ void projTransform( D3DXVECTOR3* d_meshVertexProj,
							   int*			d_silNum,
							   D3DXMATRIX*	d_matrixWorldView,
							   D3DXMATRIX*	d_matrixProj)
{
	const int idx = blockIdx.x * g_BLOCK_SIZE + threadIdx.x;

	int silVerticesNum = (*d_silNum) * 2;
 
	if(idx >= silVerticesNum)
		return;

	//Projection Transformation
	d_meshVertexProj[idx] = matrixPntMul(d_meshVertexProj[idx], d_matrixWorldView);
	d_meshVertexProj[idx] = matrixPntMul(d_meshVertexProj[idx], d_matrixProj);
}

__global__ void cullSilouette(MeshVertex* d_meshVertex,
							 WORD* d_indices,
							 int* d_maxIndiceNum,
							 D3DXVECTOR3* d_candidateSilhouetteVertex,
							 int*	d_silNum,
							 bool*	d_isSilhouette,
							 D3DXMATRIX* d_matrixWorldView)
{
	const int idx = blockIdx.x * g_BLOCK_SIZE + threadIdx.x;

	int triangleNum = *d_maxIndiceNum / 3;

	if(idx >= *d_silNum * triangleNum)
		return;

	int silIdx = idx / triangleNum;

	if(!d_isSilhouette[silIdx])
		return;

	int triangleIdx = idx % triangleNum;

	D3DXVECTOR3 endPnt1 = matrixPntMul(d_candidateSilhouetteVertex[2*silIdx], d_matrixWorldView);
	D3DXVECTOR3 endPnt2 = matrixPntMul(d_candidateSilhouetteVertex[2*silIdx+1], d_matrixWorldView);

	D3DXVECTOR3 silMidPnt = (endPnt1 + endPnt2) / 2.0f;

	WORD triangleV0Idx = d_indices[3*triangleIdx];
	WORD triangleV1Idx = d_indices[3*triangleIdx+1];
	WORD triangleV2Idx = d_indices[3*triangleIdx+2];

	D3DXVECTOR3 v0Pos = d_meshVertex[triangleV0Idx].position;
	D3DXVECTOR3 v1Pos = d_meshVertex[triangleV1Idx].position;
	D3DXVECTOR3 v2Pos = d_meshVertex[triangleV2Idx].position;

	v0Pos = matrixPntMul(v0Pos, d_matrixWorldView);
	v1Pos = matrixPntMul(v1Pos, d_matrixWorldView);
	v2Pos = matrixPntMul(v2Pos, d_matrixWorldView);

	D3DXVECTOR3 origin = D3DXVECTOR3(0,0,0);
	bool isInvisible = segmentIntersectTriangle(origin, silMidPnt, v0Pos, v1Pos, v2Pos);

	if(isInvisible)
	{
		d_isSilhouette[silIdx] = false;
	}
}

__device__ bool segmentIntersectTriangle(const D3DXVECTOR3& orig, 
										 const D3DXVECTOR3& des,
										 const D3DXVECTOR3& v0, 
										 const D3DXVECTOR3& v1, 
										 const D3DXVECTOR3& v2)
{
	float t,u,v;
	const D3DXVECTOR3 tmpDir = des - orig;
	
	D3DXVECTOR3 dir = normalize(tmpDir);

	// Find vectors for two edges sharing vert0
	D3DXVECTOR3 edge1 = v1 - v0;
	D3DXVECTOR3 edge2 = v2 - v0;

	// Begin calculating determinant - also used to calculate U parameter
	D3DXVECTOR3 pvec;
	pvec = crossProduct(dir, edge2);

	// If determinant is near zero, ray lies in plane of triangle
	float det = dotProduct(edge1, pvec);

	D3DXVECTOR3 tvec;
	if( det > 0 )
	{
		tvec = orig - v0;
	}
	else
	{
		tvec = v0 - orig;
		det = -det;
	}

	if( det < 0.0001f )
		return false;

	// Calculate U parameter and test bounds
	u = dotProduct(tvec, pvec);

	if( u < 0.0f || u > det )
		return false;

	// Prepare to test V parameter
	D3DXVECTOR3 qvec;
	qvec = crossProduct(tvec, edge1);

	// Calculate V parameter and test bounds
	v = dotProduct(dir, qvec);

	if( v < 0.0f || u + v > det )
		return false;

	// Calculate t, scale parameters, ray intersects triangle
	t = dotProduct(edge2, qvec);
	FLOAT fInvDet = 1.0f / det;
	t *= fInvDet;
	u *= fInvDet;
	v *= fInvDet;
	
	if( length(orig + t * dir) > length(des - orig) )
		return false;
	else if( fabs(length(orig + t * dir) - length(des - orig)) < 0.0001 )
		return false;
		
	return true;
}

bool cudaGetCulledDataFromGPU( bool* h_isSilhouette, int h_silNum )
{
	hipMemcpy(h_isSilhouette, d_isSilhouette,	 h_silNum * sizeof(bool), hipMemcpyDeviceToHost);

	return true;
}

bool cudaCullInit( int silNum )
{
	hipError_t err = hipSuccess;

	if(silNum > h_curMaxSilNum)
	{
		h_curMaxSilNum = silNum;

		if(d_candidateSilhouetteVertex)
			hipFree(d_candidateSilhouetteVertex);

		err = hipMalloc((void**)&d_candidateSilhouetteVertex, silNum * 2 * sizeof(D3DXVECTOR3));

		if(err != hipSuccess)
			return false;

		if(d_silNum)
			hipFree(d_silNum);

		err = hipMalloc((void**)&d_silNum, sizeof(int));

		if(err != hipSuccess)
			return false;
	}

	return true;
}

bool cudaPassCullDataToGPU( D3DXVECTOR3* h_meshVertexProj, int h_silNum )
{
	if(!cudaCullInit(h_silNum))
		return false;

	hipMemcpy(d_silNum, &h_silNum, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_candidateSilhouetteVertex, h_meshVertexProj, h_silNum * 2 * sizeof(D3DXVECTOR3), hipMemcpyHostToDevice);

	return true;
}